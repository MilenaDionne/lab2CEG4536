#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define N 8
#include <iostream>

using namespace std;

__global__ void sum_reduction(int* input)
{
    //CUDA threads --> block can be split into n threads, 
    const int number_of_blocks = threadIdx.x; //block = 1
    int number_of_threads = blockDim.x; //thread = 4, ainsi cette fonction s'ex�cutera 4 fois 

    auto step_size = 1;
    

    while (number_of_threads > 0)
    {
        if (number_of_blocks < number_of_threads)
        {
            const auto fst = number_of_blocks * step_size * 2;
            const auto snd = fst + step_size;
            input[fst] += input[snd];
        }

        step_size <<= 1;
        number_of_threads >>= 1;
    }
}

void fill_array(int* data, int size) {
    for (int i = 0; i < size; i++) {
        data[i] = i+1;
    }
}


int main()
{
    //size of memory for 8 elements 
    const int size = N * sizeof(int);

    //Tableau orginal 
    int* a, * result;
    int* d_a;


    //calculate cuda time 
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
  
    //Allocate memory 
    a = (int*)malloc(size); 
    result = (int*)malloc(size); 
    hipMalloc((void**)&d_a, size);


    //Initialize table 
    fill_array(a, N);
    
    //Transfert les donn�es de l�h�te � l�appareil
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    //Start time recording 
    hipEventRecord(start, 0);
    
    //Call kernel on 1 block with 8 thread divided by 2
    sum_reduction << <1, N/2 >> > (d_a);

    
    //hipDeviceSynchronize(); No need to synchronise kernel in this program. Can stay asynchrone 
    
    //Copy to host 
    hipMemcpy(result, d_a, sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);

    cout << "La somme est de " << result[0] << " pour " << N << " elements" << endl;
    cout << "La temps de performance Cuda pour 1 block de 8 thread est: " << time << endl; 

    //Destroy event object start and stop 
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //Free card space 
    hipFree(d_a);

    //Free memory 
    free(a);
    free(result);
    
    return 0;
}

